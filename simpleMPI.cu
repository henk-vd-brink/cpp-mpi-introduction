#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */


/* Simple example demonstrating how to use MPI with CUDA
*
*  Generate some random numbers on one node.
*  Dispatch them to all nodes.
*  Compute their square root on each node's GPU.
*  Compute the average of the results using MPI.
*
*  simpleMPI.cu: GPU part, compiled with nvcc
*/

#include <iostream>
using std::cerr;
using std::endl;

#include "simpleMPI.h"

// Very simple GPU Kernel that computes square roots of input numbers
__global__ void simpleMPIKernel(float *input, float *output)
{
    int i;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    output[tid] = input[tid];
    for(i=0;i<5000;i++) {
        output[tid] = sqrt(output[tid]);
        output[tid] = output[tid] * output[tid];
    }

}

// Initialize an array with random data (between 0 and 1)
void initData(float *data, int dataSize)
{
    for (int i = 0; i < dataSize; i++)
    {
        data[i] = (float)rand() / RAND_MAX;
    }
}

// CUDA computation on each node
// No MPI here, only CUDA
void computeGPU(float *hostData, int blockSize, int gridSize)
{
    int dataSize = blockSize * gridSize;

    // Allocate data on GPU memory
    float *deviceInputData = NULL;
    hipMalloc((void **)&deviceInputData, dataSize * sizeof(float));

    float *deviceOutputData = NULL;
    hipMalloc((void **)&deviceOutputData, dataSize * sizeof(float));

    // Copy to GPU memory
    hipMemcpy(deviceInputData, hostData, dataSize * sizeof(float), hipMemcpyHostToDevice);

    // Run kernel
    simpleMPIKernel<<<gridSize, blockSize>>>(deviceInputData, deviceOutputData);

    // Copy data back to CPU memory
    hipMemcpy(hostData, deviceOutputData, dataSize *sizeof(float), hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(deviceInputData);
    hipFree(deviceOutputData);
}

float sum(float *data, int size)
{
    float accum = 0.f;

    for (int i = 0; i < size; i++)
    {
        accum += data[i];
    }

    return accum;
}
